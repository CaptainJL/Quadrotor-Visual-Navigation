#include "hip/hip_runtime.h"
#include "gpu_w_flow.hpp"



using namespace std;
using namespace flowfilter;
using namespace flowfilter::gpu;




__global__ void wflow_K(gpuimage_t<float2> flow, gpuimage_t<float3> sflow, WFlow_Params params)
{
	int2 size = 	make_int2(flow.width,flow.height);
	int2 pix = 		make_int2(blockIdx.x*blockDim.x+threadIdx.x,
							blockIdx.y*blockDim.y+threadIdx.y);
	if(pix.x>=size.x || pix.y>=size.y){return;}

  	// Access
  	float2 flow_pix = 	*coordPitch(flow, pix);
	double flowX = 		(double)(flow_pix.x);
	double flowY = 		(double)(flow_pix.y);

	// Simplifiers
	double xd =			pix.x-params.x0;
	double yd =			pix.y-params.y0;
	double xymag = 		sqrt( xd*xd + yd*yd );
	double hypmag =		sqrt( xymag*xymag + params.f*params.f);

	// Structure Flow
  	double3 sflow_p = 	make_double3(0.0, 0.0, 0.0);

  	// printf("sfx = %0.3f\n", (float)(sflow_p.x));


	if( xymag>=10 && xymag<=params.rad )
	{
		double theta =	atan2(xymag,params.f);
		double S_area = pow(cos(theta),3)/(params.f*params.f);

		sflow_p.x = S_area/params.S_cap*params.f/xymag*flowX;
		sflow_p.y = S_area/params.S_cap*params.f/xymag*flowY;
		sflow_p.z = -S_area/params.S_cap*params.f/xymag*tan(theta)*(flowX*xd/xymag + flowY*yd/xymag);

		/*if ( (pix.x==170) && (pix.y==100) )
		{ 
			printf("sf = %0.3f, %0.3f, %0.3f\n", sflow_p.x, sflow_p.y, sflow_p.z); 
			printf("%0.3f, %0.3f, %0.3f\n", params.f, params.S_cap, theta);
		}*/

	}
	else
	{
		sflow_p.x = 0.0;
		sflow_p.y = 0.0;
		sflow_p.z = 0.0;
	}


  	float3 sflow_pix =			make_float3((float)(sflow_p.x), (float)(sflow_p.y), (float)(sflow_p.z));
	*coordPitch(sflow, pix) = 	sflow_pix;
}






void configureKernelGrid(const int height, const int width, const dim3 block, dim3& grid) 
{
    float w = width;
    float h = height;
    float x = block.x;
    float y = block.y;

    grid.x = (int)ceilf(w / x);
    grid.y = (int)ceilf(h / y);
    grid.z = 1;
}



WFlow_GPU::WFlow_GPU()
{
  	
  	

}
WFlow_GPU::~WFlow_GPU()
{

}






















/* OLD VERSION
__global__ void wflow_K(gpuimage_t<float2> flow, gpuimage_t<float3> sflow, WFlow_Params params)
{
	int2 size = 	make_int2(flow.width,flow.height);
	int2 pix = 		make_int2(blockIdx.x*blockDim.x+threadIdx.x,
							blockIdx.y*blockDim.y+threadIdx.y);
	if(pix.x>=size.x || pix.y>=size.y){return;}

  	// Access
  	float2 flow_pix = 	*coordPitch(flow, pix);
	double flowX = 		(double)(flow_pix.x);
	double flowY = 		(double)(flow_pix.y);

	// Simplifiers
	double xd =			abs(pix.x-params.x0);
	double xd2 =		xd*xd;
	double yd =			abs(pix.y-params.y0);
	double yd2 =		yd*yd;
	double f2 = 		params.f*params.f;
	double pl2 =		params.pix_len*params.pix_len;
	double dist = 		sqrt( (xd)*(xd) + (yd)*(yd) );

	// Structure Flow
  	double3 sflow_p = 	make_double3(0.0, 0.0, 0.0);

  	// printf("sfx = %0.3f\n", (float)(sflow_p.x));


	if( 78.0<=dist && dist<=params.rad )
	{
		sflow_p.x = -(1.0/(f2)*(pl2)*(flowX*(xd2/(xd2+yd2+f2)-1.0)*1.0/sqrt(xd2+yd2+f2)+flowY*(xd)*(yd)*1.0/pow(xd2+yd2+f2,3.0/2.0))*1.0/pow(1.0/(f2)*(xd2+yd2)+1.0,3.0/2.0))/params.S_cap;
	   	sflow_p.y = -(1.0/(f2)*(pl2)*1.0/pow(1.0/(f2)*(xd2+yd2)+1.0,3.0/2.0)*(flowY*(yd2/(xd2+yd2+f2)-1.0)*1.0/sqrt(xd2+yd2+f2)+flowX*(xd)*(yd)*1.0/pow(xd2+yd2+f2,3.0/2.0)))/params.S_cap;
	   	sflow_p.z = -(1.0/(f2)*(pl2)*1.0/pow(1.0/(f2)*(xd2+yd2)+1.0,3.0/2.0)*(params.f*flowX*(xd)*1.0/pow(xd2+yd2+f2,3.0/2.0)+params.f*flowY*(yd)*1.0/pow(xd2+yd2+f2,3.0/2.0)))/params.S_cap;		
	}
	else
	{
		sflow_p.x = 0.0;
		sflow_p.y = 0.0;
		sflow_p.z = 0.0;
	}



	// if ( (pix.x==50) && (pix.y==50) )
	// { printf("sfx = %0.3f\n", (float)(sflow_p.x)); }

  	float3 sflow_pix =			make_float3((float)(sflow_p.x), (float)(sflow_p.y), (float)(sflow_p.z));
	*coordPitch(sflow, pix) = 	sflow_pix;
}
*/











void 
WFlow_GPU::calc_wflow()
{
	__block = dim3(16, 16, 1);
	configureKernelGrid((int)wf_params.cam_h, (int)wf_params.cam_w, __block, __grid);

	wflow_K<<<__grid, __block>>>(flow_gpu.wrap<float2>(), sflow_gpu.wrap<float3>(), wf_params);

}
